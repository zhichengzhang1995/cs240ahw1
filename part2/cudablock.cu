#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <float.h>
#include <math.h>

#include <sys/time.h>
#include <assert.h>
#define BLOCK_SIZE 16

/* Helper functions */
double read_timer( )
{
    static bool initialized = false;
    static struct timeval start;
    struct timeval end;
    if( !initialized )
    {
        gettimeofday( &start, NULL );
        initialized = true;
    }

    gettimeofday( &end, NULL );

    return (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
}

__global__ void square_dgemm(float* devM, float* devN, float* devP, int width)
{
  __shared__ float sM[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float sN[BLOCK_SIZE][BLOCK_SIZE];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int col = bx * BLOCK_SIZE + tx;
  int row = by * BLOCK_SIZE + ty;

  float sum = 0;


  for( int i = 0; i < width / BLOCK_SIZE; i++ ){
        sM[ty][tx] = devM[row * width + (i * BLOCK_SIZE + tx)];
        sN[ty][tx] = devN[col + (i * BLOCK_SIZE + ty) * width];
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k){
                sum += sM[ty][k] * sN[k][tx];
                __syncthreads();
        }
  }

  devP[row * width + col] = sum;
}

void fill( float *p, int n){
    for (int i = 0; i < n; i++)
        p[i] = 2 * (float) drand48() - 1;
}
/* The benchmarking program */

int main( int argc, char **argv )
{
	int n = 1024;
	int m = 1024;
	int k = 1024;
	float *h_a, *h_b, *h_c;

	//hipHostMalloc((void **) &h_a, sizeof(float)*m*n);
	//hipHostMalloc((void **) &h_b, sizeof(float)*n*k);
	//hipHostMalloc((void **) &h_c, sizeof(float)*m*k);
	 h_a = (float *)malloc( n*n* sizeof(float) );
 	 h_b = (float *)malloc( n*n* sizeof(float) );
  	 h_c = (float *)malloc( n*n* sizeof(float) );

  /* random initialize matrix*/
//	for (int i= 0; i<m; ++i){
//		for (int j = 0; j<n; ++j){
//			h_a[i * n + j] = rand() % 1024;
//		}
//	}
//	for (int i = 0; i<n; ++i){
//		for (int j=0; j<k; ++j){
//			h_b[i * k +j] = rand() % 1024;
//		}
//	}

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

        int grid_rows = n / BLOCK_SIZE;
        int grid_cols = n / BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);


	float *d_a, *d_b, *d_c;
	hipMalloc((void **) &d_a, sizeof(float)*m*n);
	hipMalloc((void **) &d_b, sizeof(float)*n*k);
	hipMalloc((void **) &d_c, sizeof(float)*m*k);

	fill(h_a, n*n);
	fill(h_b, n*n);
	fill(h_c, n*n);

        double seconds_copy = read_timer();

	hipMemcpy(d_a, h_a, sizeof(float)*m*n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(float)*m*n, hipMemcpyHostToDevice);
	seconds_copy = read_timer()-seconds_copy;
	printf("cpu to device copy is %f\n",seconds_copy);
	double seconds=-1.0;
	double Gflop_s = 0.0, Gflop_s1 = 0.0;
	for (int n_iterations = 1; seconds<0.1;	n_iterations*=2){
	//warmup
	square_dgemm<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, n);

	//measure
	seconds = read_timer();
	for(int i=0; i<n_iterations;i++){
	square_dgemm<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, n);
	        seconds = read_timer()-seconds;
	}
//	seconds = read_timer()-seconds;
	Gflop_s1 = (2e-9 * n * n * n * n_iterations)/(seconds);

	}
//	double seconds_copy1 = read_timer();
	hipMemcpy(h_c, d_c, sizeof(float)*m*k, hipMemcpyDeviceToHost);
//	seconds_copy1 = read_timer()-seconds_copy1;
//	printf("device to cpu copy is %f\n", seconds_copy1);
	seconds_copy = seconds+seconds_copy;
	hipDeviceSynchronize();

	hipEventSynchronize(stop);

	Gflop_s = (Gflop_s1*seconds)/(seconds_copy);

	printf("Time of GPU cal is %f s(with copy)\n Time of GPU cal is %f s (without copy) Gflop_s with copy is %g\n Gflop_s without copy is %g\n", seconds_copy, seconds, Gflop_s, Gflop_s1);

	int all_good = 1;
	for (int i=0; i<m; ++i){
		for (int j=0; j<k; ++j){
			if (h_c[i*k+j] != h_c[i*k+j]){
				all_good = 0;
			}

		}
	}
	if (all_good){
		printf("all good!!\n");

	}
	else{
		printf("incorrect!!\n");
	}

		/*Deallocate memory*/
        hipFree( d_a );
        hipFree( d_b );
        hipFree( d_c );
//	hipHostFree(h_a);
//	hipHostFree(h_b);
//	hipHostFree(h_c);
 	free(h_a);
	free(h_b);
	free(h_c);

    return 0;
}
