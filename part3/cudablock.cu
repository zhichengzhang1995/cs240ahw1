#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#define BLOCK_SIZE 16

__global__ void square_dgemm(float* devM, float* devN, float* devP, int width)
{
  __shared__ float sM[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float sN[BLOCK_SIZE][BLOCK_SIZE];
  int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;

  float sum = 0;


  for( int i = 0; i < width / BLOCK_SIZE; i++ ){
        sM[threadIdx.y][threadIdx.x] = devM[row * width + (i * BLOCK_SIZE + threadIdx.x)];
        sN[threadIdx.y][threadIdx.x] = devN[col + (i * BLOCK_SIZE + threadIdx.y) * width];
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k){
                sum += sM[threadIdx.y][k] * sN[k][threadIdx.x];
                __syncthreads();
        }
  }

  devP[row * width + col] = sum;
}

/* Helper functions */

double read_timer( )
{
    static bool initialized = false;
    static struct timeval start;
    struct timeval end;
    if( !initialized )
    {
        gettimeofday( &start, NULL );
        initialized = true;
    }

    gettimeofday( &end, NULL );

    return (end.tv_sec - start.tv_sec) + 1.0e - 6 * (end.tv_usec - start.tv_usec);
}

void fill( float *p, int n){
    for (int i = 0; i < n; i++)
        p[i] = 2 * (float) drand48() - 1;
}
/* The benchmarking program */

int main( int argc, char **argv )
{
	int n = 1600;
	int m = 1600;
	int k = 1600;
	float *A, *B, *C;

	A = (float *)malloc( n * n * sizeof(float) );
 	B = (float *)malloc( n * n * sizeof(float) );
  C = (float *)malloc( n * n * sizeof(float) );

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  int grid_rows = n / BLOCK_SIZE;
  int grid_cols = n / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	float *d_a, *d_b, *d_c;
	hipMalloc((void **) &d_a, sizeof(float)*m*n);
	hipMalloc((void **) &d_b, sizeof(float)*n*k);
	hipMalloc((void **) &d_c, sizeof(float)*m*k);

	fill(A, n*n);
	fill(B, n*n);
	fill(C, n*n);

  double seconds_copy = read_timer();

	hipMemcpy(d_a, A, sizeof(float)*m*n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, sizeof(float)*m*n, hipMemcpyHostToDevice);
	seconds_copy = read_timer()-seconds_copy;
	printf("cpu to device copy is %f\n",seconds_copy);
	double seconds=-1.0;
	double Gflop_s = 0.0, Gflop_s1 = 0.0;
	for (int n_iterations = 1; seconds<0.1;	n_iterations*=2){
    //warmup
    square_dgemm<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, n);
    //measure
    seconds = read_timer();
  	for(int i=0; i<n_iterations;i++){
    	square_dgemm<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, n);
    	        seconds = read_timer()-seconds;
  	}
    Gflop_s1 = (2e-9 * n * n * n * n_iterations)/(seconds);
	}

	hipMemcpy(C, d_c, sizeof(float)*m*k, hipMemcpyDeviceToHost);

	seconds_copy = seconds+seconds_copy;
	hipDeviceSynchronize();
	hipEventSynchronize(stop);

	Gflop_s = (Gflop_s1*seconds)/(seconds_copy);

	printf("Time of GPU cal is %f s(with copy)\n Time of GPU cal is %f s (without copy) Gflop_s with copy is %g\n Gflop_s without copy is %g\n", seconds_copy, seconds, Gflop_s, Gflop_s1);

	int check = 0;
	for (int i=0; i<m; ++i){
		for (int j=0; j<k; ++j){
			if (C[i*k+j] != C[i*k+j]){
				all_good = 1;
			}

		}
	}
	if (all_good){
		printf("Wrong\n");

	}

	/*Deallocate memory*/
  hipFree( d_a );
  hipFree( d_b );
  hipFree( d_c );

 	free(A);
	free(B);
	free(C);

  return 0;
}
